// REQUIRES: nvptx-registered-target

// Checking to see that __nvvm_reflect resolves to the correct llvm.nvvm.reflect
// intrinsic
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -disable-llvm-passes -S -emit-llvm -x c++ %s -o - | FileCheck %s --check-prefix=NO_NVVM_REFLECT_PASS

// Prepare bitcode file to link with
// RUN: %clang_cc1 -triple nvptx-unknown-cuda -emit-llvm-bc \
// RUN:    -disable-llvm-passes -o %t.bc %s

// Checking to see if the correct values are substituted for the nvvm_reflect
// call when llvm passes are enabled.
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_50 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_1
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_52 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_2
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_53 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_3
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_60 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_4
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_61 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_5
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_62 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_6
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_70 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_7
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_72 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_8
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_75 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_9
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_80 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_10
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -target-cpu \
// RUN:    sm_86 -S -o /dev/null %s -mllvm -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=ARCH_REFLECT_11

// Check to see that nvvm_reflect("__CUDA_FTZ") returns 1 or 0 based on value
// of -fdenormal-fp-math-f32 flag
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda \
// RUN:    -fdenormal-fp-math-f32=preserve-sign -S -o /dev/null %s -mllvm \
// RUN:    -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=FTZ_REFLECT
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda \
// RUN:    -fdenormal-fp-math-f32=ieee -S -o /dev/null %s -mllvm \
// RUN:    -print-after-all 2>&1 \
// RUN:    | FileCheck %s --check-prefix=NO_FTZ_REFLECT

#include "Inputs/hip/hip_runtime.h"

__device__ int foo_arch() {
  // NO_NVVM_REFLECT_PASS: call i32 @llvm.nvvm.reflect
  // ARCH_REFLECT_1: ret i32 500
  // ARCH_REFLECT_2: ret i32 520
  // ARCH_REFLECT_3: ret i32 530
  // ARCH_REFLECT_4: ret i32 600
  // ARCH_REFLECT_5: ret i32 610
  // ARCH_REFLECT_6: ret i32 620
  // ARCH_REFLECT_7: ret i32 700
  // ARCH_REFLECT_8: ret i32 720
  // ARCH_REFLECT_9: ret i32 750
  // ARCH_REFLECT_10: ret i32 800
  // ARCH_REFLECT_11: ret i32 860
  return __nvvm_reflect("__CUDA_ARCH");
}

__device__ int foo_ftz() {
  // FTZ_REFLECT: ret i32 1
  // NO_FTZ_REFLECT: ret i32 0
  return __nvvm_reflect("__CUDA_FTZ");
}

